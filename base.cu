

#include <iostream>
#include <iomanip>
#include <hip/hip_runtime.h>

#define TILE_DIM W  // Ensure TILE_DIM is equal to W for the matrix multiplication
#define M 512  // Matrix A: number of rows
#define W 32   // Matrix A: number of columns, Matrix B: number of rows
#define N 512  // Matrix B: number of columns

void initializeMatrix(float *matrix, int rows, int cols) {
    for (int i = 0; i < rows; i++) {
        for (int j = 0; j < cols; j++) {
            matrix[i * cols + j] = static_cast<float>(rand()) / RAND_MAX;
        }
    }
}

__global__ void simpleMultiply(float *a, float* b, float *c) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;  // Calculate row index in matrix C
    int col = blockIdx.x * blockDim.x + threadIdx.x;  // Calculate column index in matrix C
    float sum = 0.0f;

    // Ensure that the thread is within matrix boundaries
    if (row < M && col < N) {
        for (int i = 0; i < W; i++) {
            sum += a[row * W + i] * b[i * N + col];  // Matrix multiplication
        }
        c[row * N + col] = sum;  // Store result in matrix C
    }
}

int main() {
    // Define matrix sizes
    int sizeA = M * W * sizeof(float);  // Size for matrix A (M x W)
    int sizeB = W * N * sizeof(float);  // Size for matrix B (W x N)
    int sizeC = M * N * sizeof(float);  // Size for result matrix C (M x N)

    // Allocate host memory
    float *h_a = (float*)malloc(sizeA);
    float *h_b = (float*)malloc(sizeB);
    float *h_c = (float*)malloc(sizeC);

    // Initialize matrices A and B
    initializeMatrix(h_a, M, W);
    initializeMatrix(h_b, W, N);

    // Allocate device memory
    float *d_a, *d_b, *d_c;
    hipMalloc((void**)&d_a, sizeA);
    hipMalloc((void**)&d_b, sizeB);
    hipMalloc((void**)&d_c, sizeC);

    // Copy matrices A and B to device
    hipMemcpy(d_a, h_a, sizeA, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, sizeB, hipMemcpyHostToDevice);

    // Define grid and block dimensions
    dim3 dimBlock(TILE_DIM, TILE_DIM);  // Each block computes a TILE_DIM x TILE_DIM tile of matrix C
    dim3 dimGrid((N + TILE_DIM - 1) / TILE_DIM, (M + TILE_DIM - 1) / TILE_DIM);  // Grid size depends on matrix size

    // Launch kernel
    simpleMultiply<<<dimGrid, dimBlock>>>(d_a, d_b, d_c);

    // Copy result matrix C back to host
    hipMemcpy(h_c, d_c, sizeC, hipMemcpyDeviceToHost);

    // Display a part of the result matrix
    std::cout << "Result matrix C (first 10 x 10 block):" << std::endl;
    std::cout << std::fixed << std::setprecision(5);
    for (int i = 0; i < 10; i++) {
        for (int j = 0; j < 10; j++) {
            std::cout << h_c[i * N + j] << " ";
        }
        std::cout << std::endl;
    }

    // Free device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    // Free host memory
    free(h_a);
    free(h_b);
    free(h_c);

    return 0;
}
