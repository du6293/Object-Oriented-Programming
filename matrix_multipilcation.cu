#include <iostream>
#include <iomanip>
#include <hip/hip_runtime.h>

#define TILE_DIM W  // Ensure TILE_DIM is equal to W for the matrix multiplication
#define M 512  // Matrix A: number of rows
#define W 32   // Matrix A: number of columns, Matrix B: number of rows
#define N 512  // Matrix B: number of columns

void initializeMatrix(float *matrix, int rows, int cols) {
    for (int i = 0; i < rows; i++) {
        for (int j = 0; j < cols; j++) {
            matrix[i * cols + j] = static_cast<float>(rand()) / RAND_MAX;
        }
    }
}

__global__ void sharedABMultiply(float *a, float *b, float *c)  // Corrected the function signature
{
    __shared__ float aTile[TILE_DIM][TILE_DIM], bTile[TILE_DIM][TILE_DIM];  // Both A and B use shared memory

    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    float sum = 0.0f;

    aTile[threadIdx.y][threadIdx.x] = a[row * W + threadIdx.x];  // Load part of matrix A into shared memory
    bTile[threadIdx.y][threadIdx.x] = b[threadIdx.y * N + col];  // Load part of matrix B into shared memory
    __syncthreads();  // Synchronize all threads within the block

    for (int i = 0; i < TILE_DIM; i++) {
        sum += aTile[threadIdx.y][i] * bTile[i][threadIdx.x];  // Both matrices use shared memory
    }
    c[row * N + col] = sum;  // Store result in matrix C
}

int main() {
    // Define matrix sizes
    int sizeA = M * W * sizeof(float);  // Size for matrix A (M x W)
    int sizeB = W * N * sizeof(float);  // Size for matrix B (W x N)
    int sizeC = M * N * sizeof(float);  // Size for result matrix C (M x N)

    // Allocate host memory
    float *h_a = (float*)malloc(sizeA);
    float *h_b = (float*)malloc(sizeB);
    float *h_c = (float*)malloc(sizeC);

    // Initialize matrices A and B
    initializeMatrix(h_a, M, W);
    initializeMatrix(h_b, W, N);

    // Allocate device memory
    float *d_a, *d_b, *d_c;
    hipMalloc((void**)&d_a, sizeA);
    hipMalloc((void**)&d_b, sizeB);
    hipMalloc((void**)&d_c, sizeC);

    // Copy matrices A and B to device
    hipMemcpy(d_a, h_a, sizeA, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, sizeB, hipMemcpyHostToDevice);

    // Define grid and block dimensions
    dim3 dimBlock(TILE_DIM, TILE_DIM);  // Each block computes a TILE_DIM x TILE_DIM tile of matrix C
    dim3 dimGrid((N + TILE_DIM - 1) / TILE_DIM, (M + TILE_DIM - 1) / TILE_DIM);  // Grid size depends on matrix size

    // Launch kernel with correct name and parameters
    sharedABMultiply<<<dimGrid, dimBlock>>>(d_a, d_b, d_c);

    // Copy result matrix C back to host
    hipMemcpy(h_c, d_c, sizeC, hipMemcpyDeviceToHost);

    // Display a part of the result matrix
    std::cout << "Result matrix C (first 10 x 10 block):" << std::endl;
    std::cout << std::fixed << std::setprecision(6);
    for (int i = 0; i < 10; i++) {
        for (int j = 0; j < 10; j++) {
            std::cout << h_c[i * N + j] << " ";
        }
        std::cout << std::endl;
    }

    // Free device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    // Free host memory
    free(h_a);
    free(h_b);
    free(h_c);

    return 0;
}

